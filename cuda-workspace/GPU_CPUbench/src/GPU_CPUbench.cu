
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 1024

__global__ void vectorMultGPU(float *a, float *b, float *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
	{
		c[i] = a[i] * b[i];
	}
}

void vectorMultCPU(float *a, float *b, float *c, int n)
{
	int i;

	for (i = 0; i < n; ++i)
	{
		c[i] = a[i] * b[i];
	}
}

int main()
{
	float *a, *b, *c, *GPUout;
	float *d_a, *d_b, *d_c;
	int i;

	time_t curTime, baseTime;

	a = (float*)malloc(SIZE*sizeof(float));
	b = (float*)malloc(SIZE*sizeof(float));
	c = (float*)malloc(SIZE*sizeof(float));
	GPUout = (float*)malloc(SIZE*sizeof(float));

	hipMalloc(&d_a, SIZE*sizeof(float));
	hipMalloc(&d_b, SIZE*sizeof(float));
	hipMalloc(&d_c, SIZE*sizeof(float));

	for(i = 0; i < SIZE; ++i)
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(float), hipMemcpyHostToDevice);

	baseTime = curTime = time(NULL);
	while(curTime < baseTime + 10) //Runs for about 10 seconds
	{
		vectorMultCPU(a, b, c, SIZE);
		vectorMultGPU<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);
		curTime = time(NULL);
	}

	hipMemcpy(GPUout, d_c, SIZE*sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < 20; ++i)
	{
		printf("CPU[%d] = %f, GPU[%d] = %f\n", i, c[i], i, GPUout[i]);
	}

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
