
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define SIZE 100000

__global__ void vectorMult(float *a, float *b, float *c, int n)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	while (i < n)
	{
		c[i] = a[i] * b[i];
		i+= blockDim.x * gridDim.x;
	}
}

int main()
{
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int i;

	time_t curTime, baseTime;

	a = (float*)malloc(SIZE*sizeof(float));
	b = (float*)malloc(SIZE*sizeof(float));
	c = (float*)malloc(SIZE*sizeof(float));

	hipMalloc(&d_a, SIZE*sizeof(float));
	hipMalloc(&d_b, SIZE*sizeof(float));
	hipMalloc(&d_c, SIZE*sizeof(float));

	for(i = 0; i < SIZE; i++)
	{
		a[i] = b[i] = (float)i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(float), hipMemcpyHostToDevice);

	int count = 0;

	baseTime = curTime = time(NULL);
	while(curTime < baseTime + 10) //Runs for about 10 seconds
	{
		count++;
		hipDeviceSynchronize();
		vectorMult<<< (SIZE+511)/512, 512 >>>(d_a, d_b, d_c, SIZE);
		curTime = time(NULL);
	}

	hipMemcpy(c, d_c, SIZE*sizeof(float), hipMemcpyDeviceToHost);

	printf("Call Count: %d\n", count);
	for (i = 0; i < 10; ++i)
	{
		printf("c[%d] = %f\n", i, c[i]);
	}
	printf("c[99,999] = %f\n", c[SIZE-1]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

